
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <fstream>
#include <vector>
#include <iostream>
#include <chrono>

/**
 * NAVIER STOKES COMPUTATIONAL MODULE
 * using SHARED MEMORY & GLOBAL MEMORY
*/

void execGM(int nx, int ny);

using namespace std;
typedef vector<vector<float>> matrix;

__host__ void cudaErrorCheck(hipError_t err)
{
    if (err != hipSuccess)
        std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl << std::flush;
}

__global__ void updateUV(
  int N,
  int nx,
  int ny,
  float dt,
  float dx,
  float dy,
  float dx2,
  float dy2,
  float nu,
  float rho,
  float *u,
  float *v,
  float *p
)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int tid = j*nx+i;
  if (i < nx && j <ny)
  {
    extern __shared__ float u_v_n[];
    // load u_n and v_n into shared mem into on 1D array
    u_v_n[tid] = u[tid];
    u_v_n[tid+N] = v[tid];
    float newU = 0.0;
    float newV = 0.0;
    __syncthreads();

    
    if (i > 0 && j > 0 && i< nx-1 && j <ny-1)
    {
      float u_ij = u_v_n[tid];
      float v_ij = u_v_n[tid+N];

      int jp1_i = (j+1)*nx+i;
      float v_jp1_i = u_v_n[jp1_i+N];
      float u_jp1_i = u_v_n[jp1_i];

      int jm1_i = (j-1)*nx+i;
      float v_jm1_i = u_v_n[jm1_i+N];
      float u_jm1_i = u_v_n[jm1_i];

      int j_ip1 = j*nx+i+1;
      float u_j_ip1 = u_v_n[j_ip1];
      float v_j_ip1 = u_v_n[j_ip1+N];

      int j_im1 = j*nx+i-1;
      float u_j_im1 = u_v_n[j_im1];
      float v_j_im1 = u_v_n[j_im1+N];

      newU = u_ij - u_ij * dt / dx * (u_ij - u_j_im1)
                    - u_ij * dt / dy * (u_ij - u_jm1_i)
                    - dt / (2 * rho * dx) * (p[j*nx+i+1] - p[j*nx+i-1])
                    + nu * dt / dx2 * (u_j_ip1 - 2 * u_ij + u_j_im1)
                    + nu * dt / dy2 * (u_jp1_i - 2 * u_ij + u_jm1_i);

      newV = v_ij - v_ij * dt / dx * (v_ij - v_j_im1)
                    - v_ij * dt / dy * (v_ij - v_jm1_i)
                    - dt / (2 * rho * dy) * (p[(j+1)*nx+i] - p[(j-1)*nx+i])
                    + nu * dt / dx2 * (v_j_ip1 - 2 * v_ij + v_j_im1)
                    + nu * dt / dy2 * (v_jp1_i - 2 * v_ij + v_jm1_i);
    }
    __syncthreads();

    // UPDATE BOUNDARIES. Lines first and then columns.

    // u[:,-1]=0.0 v[:,-1]=0.0
    if (i==(nx-1))
    {
      newU = 0.0;
      newV = 0.0;
    }
    __syncthreads();
    
    // u[:,0]=0.0 v[:,0]=0.0

    if (i==0)
    {
      newU = 0.0;
      newV = 0.0;
    }
    __syncthreads();

    //  u[0, :]  = 0 , v[0, :]  = 0
    if (j==0)
    {
      newU = 0.0;
      newV = 0.0;
    }
    __syncthreads();
    //  u[-1, :] = 1 , v[-1, :]  = 0
    if (j==(ny-1))
    {
      newU = 1.0;
      newV = 0.0;
    }
    __syncthreads();

    // Final step: write into GM
    u[tid] = newU;
    v[tid] = newV;
  }
}

__global__ void updateP(
  int N, 
  int nit,
  int nx, 
  int ny,
  float dx2,
  float dy2,
  float *p,
  float *b
  )
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int tid = j*nx+i;
  if (i < nx && j <ny)
  {
    extern __shared__ float tempP[];
    // 2 copies of p: tempP[0..N-1] is p and tempP[N..N*N-1] is p_n
    tempP[tid] = p[tid];
    int tid_pn=tid+N;
    tempP[tid_pn] = p[tid];
    __syncthreads();

    float b_ij_c = 0.0;
    float sum2_dx_dy = 0.0;
    if (i>0 && j>0 && i < (nx-1) && j <(ny-1))
    {
      b_ij_c = b[tid] * dx2 * dy2;
      sum2_dx_dy =  2*(dx2 + dy2);
    }
  
    
    for (int it=0; it<nit; it++)
    {
      tempP[tid_pn]=tempP[tid];
      __syncthreads();

      if (i>0 && j>0 && i < (nx-1) && j <(ny-1))
      {
        // updating p[j, i]
        // if (j==4 && i==2) printf("res1=%.10f, res2 = %.10f, bc =%.10f", dy2*(tempP[tid_pn+1] + tempP[tid_pn-1]), dx2*(tempP[tid_pn+nx] + tempP[tid_pn-nx]), b_ij_c);
        tempP[tid] = (dy2*(tempP[tid_pn+1] + tempP[tid_pn-1]) + dx2*(tempP[tid_pn+nx] + tempP[tid_pn-nx]) - b_ij_c)/sum2_dx_dy;

      }
      __syncthreads();

       // p[:, -1] = p[:, -2]
      if (i==(nx-1))
      {
        tempP[tid] = tempP[tid-1];
      }
      __syncthreads();
      
      
      // p[0, :] = p[1, :]
      if (j==0)
      {
        tempP[tid] = tempP[tid+nx];
      }
      __syncthreads();

     

      //p[:, 0] = p[:, 1]
      if (i==0)
      {
        tempP[tid] = tempP[tid+1];
      }
      __syncthreads();

      //  p[-1, :] = 0
      if (j==(ny-1))
      {
        tempP[tid] = 0.0;
      }
      __syncthreads();
    }

    // end: copy shared array 'p' into global memory array 'p'
    p[tid] = tempP[tid];
  }
}

__global__ void computeB(
  int nx, 
  int ny, 
  float rho,
  float dt,
  float dx,
  float dy,
  float *u, 
  float *v,
  float *b
  )
{
   int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;

  if (i > 0 && j > 0 && i< nx-1 && j <ny-1)
  {
    int tid = j*nx+i;
    int jp1_i = tid+nx;
    float v_jp1_i = v[jp1_i];
    float u_jp1_i = u[jp1_i];

    int jm1_i = tid-nx;
    float v_jm1_i = v[jm1_i];
    float u_jm1_i = u[jm1_i];

    int j_ip1 = tid+1;
    float u_j_ip1 = u[j_ip1];
    float v_j_ip1 = v[j_ip1];

    int j_im1 = tid-1;
    float u_j_im1 = u[j_im1];
    float v_j_im1 = v[j_im1];

    // b[j, i]
    float u_j_di = (u_j_ip1 - u_j_im1) / (2 * dx);
    float v_j_di = (v_jp1_i - v_jm1_i) / (2 * dy);
    b[tid] = rho * (
      1 / dt * ((u_j_ip1 - u_j_im1) / (2 * dx) + (v_jp1_i - v_jm1_i) / (2 * dy)) 
      - u_j_di*u_j_di 
      - ((u_jp1_i - u_jm1_i) * (v_j_ip1 - v_j_im1) / (2 * dy * dx)) 
      - v_j_di*v_j_di
    );
  }
}


__global__ void init_arrays(
  int maxTid, 
  int nx, 
  float *u, 
  float *v, 
  float *p
  )
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int tid = j*nx+i;
  if (tid < maxTid)
  {
    u[tid] = 0.0;
    v[tid] = 0.0;
    p[tid] = 0.0;
  }
}

int main() {
    printf("NAVIER STOKES FOR A 32x32 GRID -------->> EXECUTE: 32x32_plot.py // COMPARISON: 32x32_cavity.py\n");


  auto start = std::chrono::high_resolution_clock::now();

  int nx = 32;
  int ny = 32;
  int nt = 500;
  int nit = 50;
  double dx = 2. / (nx - 1);
  double dx2 = dx*dx;
  double dy = 2. / (ny - 1);
  double dy2 = dy*dy;
  double dt = .01;
  double rho = 1.;
  double nu = .02;

  int nbElements = nx*ny;

  float *u;
  float *v;
  float *p;
  float *b;
  float *un;
  float *vn;

  cudaErrorCheck(hipMallocManaged(&u, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&v, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&b, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&p, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&un, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&vn, sizeof(float)*nx*ny));

  dim3 dimGrid((nx+31)/32, (ny+31)/32, 1);
  dim3 dimBlock(32, 32, 1);

  // initializing u, v, p to 0.0
  init_arrays<<<dimGrid, dimBlock>>>(nbElements, nx, u, v, p);


  ofstream ufile("u.dat");
  ofstream vfile("v.dat");
  ofstream pfile("p.dat");
  for (int n=0; n<nt; n++) {

    computeB<<< dim3((nx-2+31)/32,(ny-2+31)/32,1), dim3(32, 32, 1)>>>(
      nx,
      ny,
      rho,
      dt,
      dx,
      dy,
      u,
      v,
      b
    );

    cudaErrorCheck(hipDeviceSynchronize());


    updateP<<<dimGrid, dimBlock, 2*nbElements*sizeof(float)>>>(
      nbElements,
      nit,
      nx,
      ny,
      dx2,
      dy2,
      p,
      b
    );
    cudaErrorCheck(hipDeviceSynchronize());



    updateUV<<<dimGrid, dimBlock, 2*nbElements*sizeof(float)>>>(
      nbElements,
      nx,
      ny,
      dt,
      dx,
      dy,
      dx2,
      dy2,
      nu,
      rho,
      u,
      v,
      p
    );
    cudaErrorCheck(hipDeviceSynchronize());


    if (n%10==0) {
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          ufile << u[j*nx+i] << " ";
      ufile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          vfile << v[j*nx+i] << " ";
      vfile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          pfile << p[j*nx+i] << " ";
      pfile << "\n";
    }
  }
  ufile.close();
  vfile.close();
  pfile.close();
  
  std::chrono::duration<double> total = std::chrono::high_resolution_clock::now() - start;

  start = std::chrono::high_resolution_clock::now();

  execGM(nx, ny);

  std::chrono::duration<double> totalGM = std::chrono::high_resolution_clock::now() - start;
  printf("\nSM = %fs and with only GM: %fs \n", total.count(), totalGM.count());
  remove("utemp");
  remove("vtemp");
  remove("ptemp");
}
















/*---------------------------------------------------------------------------------------------------------------------------------------*/

/**
 * 
 * THIS PART IS JUST A COPY OF THE 41X41_CAVITY.CU FILE SO THAT I CAN COMPARE
 * THE TWO METHODS WITH GLOBAL MEMORY AND SHARED MEMORY 
 * 
 * 
 */

__global__ void updateP_Cols(
  int nx,
  int offsetLeft,
  float *p
)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if (tid < nx)
  {
    p[tid] = p[tid+nx];
    p[tid+offsetLeft] = 0.0;
  }
}

__global__ void updateP_Lines(
  int maxTid,
  int nx,
  int offsetBottom,
  float *p
)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  if (j < maxTid)
  { 
    int column = j*nx;
    p[column+offsetBottom] = p[column+offsetBottom-1]; // p[:, -1] = p[:, -2]
    p[column] = p[column+1]; // p[:, 0] = p[:, 1]
  }
}

__global__ void updateUV_RLBounds(
  int nx,
  int offsetLeft,
  float *u,
  float *v
)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if (tid < nx)
  {
    u[tid] = 0.0;
    v[tid] = 0.0;
    u[tid+offsetLeft] = 1.0;
    v[tid+offsetLeft] = 0.0;
  }
}

__global__ void updateUV_TBBounds(
  int ny,
  int nx,
  int offsetBottom,
  float *u,
  float *v
)
{
  int j = threadIdx.x + blockDim.x*blockIdx.x;
  if (j < ny)
  { 
    int column = j*nx;
    u[column] = 0.0;
    v[column] = 0.0;
    u[column+offsetBottom] = 0.0;
    v[column+offsetBottom] = 0.0;
  }
}

__global__ void updateUV_GM(
  int nx,
  int ny,
  float dt,
  float dx,
  float dy,
  float dx2,
  float dy2,
  float nu,
  float rho,
  float *u,
  float *v,
  float *un,
  float *vn,
  float *p

)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;

  if (i > 0 && j > 0 && i< nx-1 && j <ny-1)
  {
    int tid = j*nx+i;

    float u_ij = un[tid];
    float v_ij = vn[tid];

    int jp1_i = (j+1)*nx+i;
    float v_jp1_i = vn[jp1_i];
    float u_jp1_i = un[jp1_i];

    int jm1_i = (j-1)*nx+i;
    float v_jm1_i = vn[jm1_i];
    float u_jm1_i = un[jm1_i];

    int j_ip1 = j*nx+i+1;
    float u_j_ip1 = un[j_ip1];
    float v_j_ip1 = vn[j_ip1];

    int j_im1 = j*nx+i-1;
    float u_j_im1 = un[j_im1];
    float v_j_im1 = vn[j_im1];

    u[tid] = u_ij - u_ij * dt / dx * (u_ij - u_j_im1)
                  - u_ij * dt / dy * (u_ij - u_jm1_i)
                  - dt / (2 * rho * dx) * (p[j*nx+i+1] - p[j*nx+i-1])
                  + nu * dt / dx2 * (u_j_ip1 - 2 * u_ij + u_j_im1)
                  + nu * dt / dy2 * (u_jp1_i - 2 * u_ij + u_jm1_i);

    v[tid] = v_ij - v_ij * dt / dx * (v_ij - v_j_im1)
                  - v_ij * dt / dy * (v_ij - v_jm1_i)
                  - dt / (2 * rho * dy) * (p[(j+1)*nx+i] - p[(j-1)*nx+i])
                  + nu * dt / dx2 * (v_j_ip1 - 2 * v_ij + v_j_im1)
                  + nu * dt / dy2 * (v_jp1_i - 2 * v_ij + v_jm1_i);

  }
}

__global__ void copyUnVn(
  int maxTid,
  int nx,
  float *un,
  float *u,
  float *vn,
  float *v
)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int tid = j*nx+i;

  if (tid < maxTid)
  {
    un[tid] = u[tid];
    vn[tid] = v[tid];
  }
}

__global__ void copyPN(
  int N, 
  int nx, 
  float *p,
  float *pn
)
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int tid = j*nx+i;
  if (tid<N)
  {
    //copy
    pn[tid] = p[tid];
  }
}
__global__ void updateP_GM(
  int nx, 
  int ny,
  float dx2,
  float dy2,
  float *p,
  float *pn,
  float *b
  )
{
  int i = threadIdx.x + blockDim.x*blockIdx.x;
  int j = threadIdx.y + blockDim.y*blockIdx.y;
  int tid = j*nx+i;
  if (i>0 && j>0 && i < (nx-1) && j <(ny-1))
  {
    p[tid] = (dy2*(pn[tid+1] + pn[tid-1]) + dx2*(pn[tid+nx] + pn[tid-nx]) - b[tid] * dx2 * dy2)/(2*(dx2 + dy2));

  }
}


void execGM(int nx, int ny)
{

  int nt = 500;
  int nit = 50;
  double dx = 2. / (nx - 1);
  double dx2 = dx*dx;
  double dy = 2. / (ny - 1);
  double dy2 = dy*dy;
  double dt = .01;
  double rho = 1.;
  double nu = .02;

  int nbElements = nx*ny;

  float *u;
  float *v;
  float *p;
  float *pn;
  float *b;
  float *un;
  float *vn;

  cudaErrorCheck(hipMallocManaged(&u, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&v, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&b, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&p, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&pn, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&un, sizeof(float)*nx*ny));
  cudaErrorCheck(hipMallocManaged(&vn, sizeof(float)*nx*ny));

  dim3 dimGrid((nx+31)/32, (ny+31)/32, 1);
  dim3 dimBlock(32, 32, 1);

  ofstream ufile("tempu.dat");
  ofstream vfile("tempv.dat");
  ofstream pfile("tempp.dat");

  // initializing u, v, p to 0.0
  init_arrays<<<dimGrid, dimBlock>>>(nbElements, nx, u, v, p);

  for (int n=0; n<nt; n++) {

    computeB<<< dim3((nx-2+31)/32,(ny-2+31)/32,1), dim3(32, 32, 1)>>>(
      nx,
      ny,
      rho,
      dt,
      dx,
      dy,
      u,
      v,
      b
    );

    cudaErrorCheck(hipDeviceSynchronize());

    for (int it =0; it<nit; ++it)
    {

      copyPN<<<dimGrid, dimBlock, 2*nbElements*sizeof(float)>>>(
        nbElements,
        nx,
        p,
        pn
      );
      cudaErrorCheck(hipDeviceSynchronize());

      updateP_GM<<<dimGrid, dimBlock, 2*nbElements*sizeof(float)>>>(
        nx,
        ny,
        dx2,
        dy2,
        p,
        pn,
        b
      );
      cudaErrorCheck(hipDeviceSynchronize());

      updateP_Lines<<<dim3((ny+31)/32, 1, 1), dim3(32, 1, 1)>>>(
      ny,
      nx,
      ny-1,
      p);
      cudaErrorCheck(hipDeviceSynchronize());

      updateP_Cols<<<dim3((nx+31)/32, 1, 1), dim3(32, 1, 1)>>>(
        nx,
        nx*(ny-1),
        p
      );
      cudaErrorCheck(hipDeviceSynchronize());
    }


    copyUnVn<<<dimGrid, dimBlock>>>(
      nbElements,
      nx,
      un,
      u,
      vn,
      v
    );
    cudaErrorCheck(hipDeviceSynchronize());


    updateUV_GM<<<dimGrid, dimBlock>>>(
      nx,
      ny,
      dt,
      dx,
      dy,
      dx2,
      dy2,
      nu,
      rho,
      u,
      v,
      un,
      vn,
      p
    );
    cudaErrorCheck(hipDeviceSynchronize());

   

    updateUV_TBBounds<<<dim3((ny+31)/32, 1, 1), dim3(32, 1, 1)>>>(
      ny,
      nx,
      ny-1,
      u,
      v
    );
    cudaErrorCheck(hipDeviceSynchronize());

    updateUV_RLBounds<<<dim3((nx+31)/32, 1, 1), dim3(32, 1, 1)>>>(
      nx,
      nx*(ny-1),
      u,
      v
    );
    cudaErrorCheck(hipDeviceSynchronize());
    if (n%10==0) {
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          ufile << u[j*nx+i] << " ";
      ufile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          vfile << v[j*nx+i] << " ";
      vfile << "\n";
      for (int j=0; j<ny; j++)
        for (int i=0; i<nx; i++)
          pfile << p[j*nx+i] << " ";
      pfile << "\n";
    }
    }

  remove("tempu.dat");
  remove("tempv.dat");
  remove("tempp.dat");

}