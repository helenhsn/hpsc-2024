
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <chrono>

void print_vec(int *vec, int size);

__global__ void fill_bucket(int maxTid, int *bucket, int *keys)
{

  // initializing shared mem within block
  extern __shared__ int temp[];
  __syncthreads();
  temp[threadIdx.x] = 0;
  __syncthreads();
  // printf("ok, blockIdx=%i, tid = %i && threadIDx.X = %i\n", blockIdx.x, tid, threadIdx.x);

  // filling the bucket -> each thread can process several elements in the key array

  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < maxTid)
  {
    int el = keys[tid];
    int old = atomicAdd(&temp[el], 1);
  }
  __syncthreads();


  // the number of threads per block is equal to the range of the bucket
  atomicAdd(&bucket[threadIdx.x], temp[threadIdx.x]); 
}


// prefix sum algorithm to fill the offset buffer
__global__ void fill_offset(int n, int *bucket, int *offset)
{
  extern __shared__ int temp[];
  int loc_tid = threadIdx.x;
  temp[loc_tid] = loc_tid>0 ? bucket[loc_tid - 1] : 0;
  __syncthreads();

  // printf("tid = %i && temp = %i\n", loc_tid, temp[loc_tid]);

  int outbuff = 0;

  for (int k=1; k<n; k<<=1)
  {
    // swap in and out buffers
    outbuff = 1 - outbuff;

    int inTID = outbuff*n+loc_tid;
    int outTID = (1 - outbuff)*n+loc_tid;
    if (loc_tid >= k)
    {
      int old = temp[inTID];
      temp[inTID] = temp[outTID] + temp[outTID-k];
      // printf("k = %i intTID %i  & value = %i , old = %i // outTID %i & value = %i // loc tid exec= %i\n",k, inTID, temp[inTID], old, outTID-k, temp[outTID-k], loc_tid);

    }
    else // simple copy
    {
      temp[inTID] = temp[outTID];
      // printf("kkk = %i intTID %i  & value = %i // outTID %i & value = %i // loc tid exec= %i\n",k, inTID, temp[inTID], outTID, temp[(1 - outbuff)*n+loc_tid], loc_tid);
    }
    __syncthreads();
    // if (loc_tid==0) printf("\n");
  }

  offset[loc_tid] = temp[outbuff*n+loc_tid];

}

__global__ void fill_key(int maxTid, int *bucket, int *offset, int *key)
{

  int tid = threadIdx.x + blockIdx.x*blockIdx.x;
  if (tid < maxTid)
  {
    int nb_occurences = bucket[tid];
    int off = offset[tid];

    for (int i=0; i<nb_occurences; i++)
    {
      key[i+off] = tid;
    }
  }
}


void bucket_sort_CPU(int *key, int *output, int n, int range)
{
  std::vector<int> bucket(range);

  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }

  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      output[j++] = i;
    }
  }

}

void print_vec(int *vec, int size)
{
  for (int i=0; i<size; i++) {
    printf("%d ",vec[i]);
  }
  printf("\n");
}





int main() {
  const int n = 1000000;
  const int range = 53;

  // initializing all the buffers
  int *key;
  int *bucket; 
  int *offset; 

  // int blockSize = (range+range%32);

  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  hipMallocManaged(&offset, range*sizeof(int));

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;

  }
  printf("\n");


  // CPU

  int outputCPU[n];


  auto start = std::chrono::steady_clock::now();

  bucket_sort_CPU(key, outputCPU, n, range);


  float CPU = (std::chrono::steady_clock::now() - start).count();

  // printf("output cpu = \n");
  // print_vec(outputCPU, n);
  // printf("\n");
 

  start = std::chrono::steady_clock::now();

  
  fill_bucket<<<(n+range-1)/range, range, range*sizeof(int)>>>(n, bucket, key);
  hipDeviceSynchronize();

  // printf("\n bucket buff = \n");
  // print_vec(bucket, range);
  
  fill_offset<<<1, range, 2*range>>>(range, bucket, offset);
  hipDeviceSynchronize();


  // printf("\n offset buff = \n");
  // print_vec(offset, range);

  fill_key<<<(range+31)/32, 32>>>(range, bucket, offset, key);
  hipDeviceSynchronize();

  // printf("\noutput gpu = \n");
  // print_vec(key, n);
  // printf("\n");

  float GPU = (std::chrono::steady_clock::now() - start).count();

  printf("Time taken by CPU VS GPU: \n%f \n%f",  CPU, GPU);

}